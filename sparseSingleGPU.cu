#include "hip/hip_runtime.h"
#include "sparseSingleGPU.cuh"

#include <algorithm>


sparseSingleGPU::sparseSingleGPU(const mxArray *sparseDouble) 
{
    
    if (!mxIsSparse(sparseDouble))
    {
        mexErrMsgIdAndTxt("MATLAB:sparseInternalOutput:invalidInputType",
                          "First argument must be sparse.");
    }

    mwIndex *ir, *jc; // ir: row indec, jc: encode row index and values in pr per coloumn
    double *pr; //value pointer
    
    // Get the starting pointer of all three data arrays.
    pr = mxGetPr(sparseDouble);     // row index array
    ir = mxGetIr(sparseDouble);     // row index array
    jc = mxGetJc(sparseDouble);     // column encrypt array
    mwSize nCols = mxGetN(sparseDouble);       // number of columns
    mwSize nRows = mxGetM(sparseDouble);       // number of rows

    // nnz = mxGetNzmax(prhs[0]); // number of possible non zero elements
    mwSize nnz = jc[nCols]; // number of non zero elements currently stored inside the sparse matrix

    #ifdef NDEBUG
        mexPrintf("Creating Sparse matrix of size %dx%d (nnz=%d)\n",nRows,nCols,nnz);
    #endif

    val_t* pr_d;
    index_t* ir_d;
    index_t* jc_d;
    
    //Create the CUDA Sparse Matrix        
    try {        
        //Create the sparse arrays on the GPU
        
        /*
        const mwSize ndims = 1; 
        //For now, use mxGPUArrays. We will only need this finally if we want to expose these arrays to Matlab. Alternatively, one could also expose them only on demand?
        mxGPUArray* d_pr = mxGPUCreateGPUArray(ndims, nnz, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
        mxGPUArray* d_ir = mxGPUCreateGPUArray(ndims, nnz, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
        mxGPUArray* d_jc = mxGPUCreateGPUArray(ndims, nnz, mxINT32_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
        */

        hipEvent_t event_start, event_start2, event_copyVal, event_copyIx, event_copyCols, event_createHandle, event_createMat;

        CHECK_CUDA(hipEventCreate(&event_start));
        CHECK_CUDA(hipEventCreate(&event_start2));
        CHECK_CUDA(hipEventCreate(&event_copyVal));
        CHECK_CUDA(hipEventCreate(&event_copyIx));
        CHECK_CUDA(hipEventCreate(&event_copyCols));
        CHECK_CUDA(hipEventCreate(&event_createHandle));
        CHECK_CUDA(hipEventCreate(&event_createMat));

        CHECK_CUDA(hipEventRecord(event_start,0));

        hipStream_t stream1, stream2, stream3;
        CHECK_CUDA(hipStreamCreate(&stream1));
        CHECK_CUDA(hipStreamCreate(&stream2));
        CHECK_CUDA(hipStreamCreate(&stream3));
        
        #pragma omp parallel sections
        {
            #pragma omp section
            {
                val_t* castedHost = this->cudaMallocAndMemcpyToDeviceWithCast<val_t,double>(&pr_d, pr, nnz, stream1);
                CHECK_CUDA(hipEventRecord(event_copyVal,stream1));
                CHECK_CUDA(hipStreamSynchronize(stream1));
                if (castedHost)
                    delete castedHost;
            }

            #pragma omp section
            {
                index_t* castedHost = this->cudaMallocAndMemcpyToDeviceWithCast<index_t,mwIndex>(&ir_d, ir, nnz, stream2);
                CHECK_CUDA(hipEventRecord(event_copyIx,stream2));
                CHECK_CUDA(hipStreamSynchronize(stream2));
                if (castedHost)
                    delete castedHost;
            }

            #pragma omp section
            {
                index_t* castedHost = this->cudaMallocAndMemcpyToDeviceWithCast<index_t,mwIndex>(&jc_d, jc, mwSize(nCols+1), stream3);
                CHECK_CUDA(hipEventRecord(event_copyCols,stream3));
                CHECK_CUDA(hipStreamSynchronize(stream3));
                if (castedHost)
                    delete castedHost;
            }       
        }
        
        hipsparseHandle_t cuSparseHandle;
        
        CHECK_CUDA(hipEventRecord(event_start2,0));  
        CHECK_CUSPARSE(hipsparseCreate(&cuSparseHandle));
        CHECK_CUDA(hipEventRecord(event_createHandle,0));       
        

        #ifdef NDEBUG
            mexPrintf("Device Pointers:\n\t%d\n\t%d\n\t%d\n",pr_d,ir_d,jc_d);
        #endif

        //this->cudaSpMatrix = std::make_shared<hipsparseSpMatDescr_t>();
        
        CHECK_CUDA(hipDeviceSynchronize());
        
        hipsparseSpMatDescr_t cuSparseMatrix;        

        //TODO match CUDA types to index_t and val_t
        //the hipsparseCreateCsc function only became available later in CUDA toolkit 11. We can though initialize a CSR matrix with similar storage pattern and say our matrix is transposed        
        #ifdef CUDA_SPMAT_CSR
            CHECK_CUSPARSE(hipsparseCreateCsr(&cuSparseMatrix, nCols, nRows, nnz, (void*) jc_d, (void*) ir_d, (void*) pr_d, cusparseType<index_t>::kind, cusparseType<index_t>::kind, HIPSPARSE_INDEX_BASE_ZERO, cudaType<val_t>::kind));
        #else
            CHECK_CUSPARSE(hipsparseCreateCsc(&cuSparseMatrix, nRows, nCols, nnz, (void*) jc_d, (void*) ir_d, (void*) pr_d, cusparseType<index_t>::kind, cusparseType<index_t>::kind, HIPSPARSE_INDEX_BASE_ZERO, cudaType<val_t>::kind));
        #endif

        CHECK_CUDA(hipEventRecord(event_createMat,0));
        CHECK_CUDA(hipEventSynchronize(event_createMat));

        this->cudaSpMatrix = std::make_shared<sparseSingleGPUdata>(pr_d, ir_d, jc_d, nRows, nCols, nnz, cuSparseHandle, cuSparseMatrix);        

        float time_copyVal, time_copyIx, time_copyCols, time_createHandle, time_createMat, timeAll;
        CHECK_CUDA(hipEventElapsedTime(&time_copyVal,event_start,event_copyVal));
        CHECK_CUDA(hipEventElapsedTime(&time_copyIx,event_start,event_copyIx));
        CHECK_CUDA(hipEventElapsedTime(&time_copyCols,event_start,event_copyCols));
        CHECK_CUDA(hipEventElapsedTime(&time_createHandle,event_start2,event_createHandle));
        CHECK_CUDA(hipEventElapsedTime(&time_createMat,event_createHandle,event_createMat));
        CHECK_CUDA(hipEventElapsedTime(&timeAll,event_start,event_createMat));

        mexPrintf("GPU Times Construct (%3.1f ms):\n\tcopy values (async): %3.1f ms\n\tcopy indices (async): %3.1f ms\n\tcopy col starts (async): %3.1f ms\n\tcusparse Initialization: %3.1f ms\n\tmatrix creation: %3.1f ms\n",
            timeAll,time_copyVal,time_copyIx,time_copyCols,time_createHandle,time_createMat);

    }
    catch (...) {
        mexErrMsgIdAndTxt("MATLAB:sparseInternalOutput:invalidInputType","CUDA sparse matrix could not be constructed!");
    }

    // no need to free memory because matlab should handle memory management of return values
}

sparseSingleGPU::~sparseSingleGPU()
{
    #ifdef NDEBUG 
        mexPrintf("Calling destructor! We still have %d references to the data object!\n",this->cudaSpMatrix.use_count() - 1);
    #endif
}

//// Getters & Setters ////
mwSize sparseSingleGPU::getNnz() const {
    return this->cudaSpMatrix->nnz;
}

mwSize sparseSingleGPU::getCols() const {
    if (this->transposed)
        return this->cudaSpMatrix->nRows;
    else
        return this->cudaSpMatrix->nCols;
}

mwSize sparseSingleGPU::getRows() const {
    if (this->transposed)
        return this->cudaSpMatrix->nCols;
    else
        return this->cudaSpMatrix->nRows;
}

mxArray* sparseSingleGPU::timesVec(const mxSingle* vals,mwSize n, bool transposeInPlace) const 
{
    //CUDA performance
    hipEvent_t event1_start, event2_prepared, event3_spmv, event4_result, event5_cleanup;
    CHECK_CUDA(hipEventCreate(&event1_start));
    CHECK_CUDA(hipEventCreate(&event2_prepared));
    CHECK_CUDA(hipEventCreate(&event3_spmv));
    CHECK_CUDA(hipEventCreate(&event4_result));
    CHECK_CUDA(hipEventCreate(&event5_cleanup));

    //Memory allocation & copy
    CHECK_CUDA(hipEventRecord(event1_start,0));

    hipStream_t stream1, stream2, stream3;
    CHECK_CUDA(hipStreamCreate(&stream1));
    CHECK_CUDA(hipStreamCreate(&stream2));
    CHECK_CUDA(hipStreamCreate(&stream3));

    //Allocate & copy multiplication vector on stream 1
    val_t* vals_d = nullptr;
    val_t* vals_h_casted = this->cudaMallocAndMemcpyToDeviceWithCast<val_t,const mxSingle>(&vals_d,vals,n,stream1);    

    //Prepare and allocate result vector on stream 2
    mwSize m;
    if (transposeInPlace)
        m = this->getCols();
    else
        m = this->getRows();
    
    mxArray* result = mxCreateNumericMatrix(m,1,mxSINGLE_CLASS,mxREAL);
    mxSingle* result_data = mxGetSingles(result);

    val_t* result_d = nullptr;
    CHECK_CUDA(hipMallocAsync((void**)&result_d, m * sizeof(val_t),stream2));
    
    // CUSPARSE SpMV: Y=α*op(A)⋅X+β*Y
    //Prepare data structure on host and stream 3

    //XOR to evaluate if we should transpose
    transposeInPlace = (transposeInPlace != this->transposed);

    #ifdef CUDA_SPMAT_CSR
        //note that the op (transpose) is the other way round due to storing the matrix as transposed CSR on the device
        hipsparseOperation_t op_transpose = transposeInPlace ? HIPSPARSE_OPERATION_NON_TRANSPOSE : HIPSPARSE_OPERATION_TRANSPOSE; 
    #else
        hipsparseOperation_t op_transpose = transposeInPlace ? HIPSPARSE_OPERATION_TRANSPOSE: HIPSPARSE_OPERATION_NON_TRANSPOSE; 
    #endif

    hipsparseDnVecDescr_t vecX, vecY;
    void* d_buffer = NULL;
    size_t bufferSize = 0;
    val_t alpha = 1.0f;
    val_t beta = 0.0f;

    hipsparseSpMVAlg_t algorithm = HIPSPARSE_SPMV_ALG_DEFAULT; //Corresponds to HIPSPARSE_SPMV_CSR_ALG1
    //hipsparseSpMVAlg_t algorithm = HIPSPARSE_SPMV_CSR_ALG1; //Not deterministic for similar input (fastest)
    //hipsparseSpMVAlg_t algorithm = HIPSPARSE_SPMV_CSR_ALG2; //Deterministic version for similar input (slower)

    //CHECK_CUSPARSE(hipsparseCreate(&cuSparseHandle)); //Now done in the constructor

    CHECK_CUSPARSE(hipsparseSetStream(this->cudaSpMatrix->cuSparseHandle,stream3));

    // create dense vector x for op(A)*x
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, n, vals_d, cudaType<val_t>::kind) );

    // create dense output vector y
    CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, m, result_d, cudaType<val_t>::kind));

    // create workspace buffer if needed
    CHECK_CUSPARSE(
        hipsparseSpMV_bufferSize(
            this->cudaSpMatrix->cuSparseHandle, op_transpose,
            &alpha, this->cudaSpMatrix->cuSparseMatrix, vecX, &beta, vecY,cudaType<val_t>::kind,
            algorithm, &bufferSize));
    if (bufferSize > 0)
        CHECK_CUDA(hipMallocAsync(&d_buffer, bufferSize,stream3));     
    
    CHECK_CUDA(hipDeviceSynchronize());
    CHECK_CUDA(hipEventRecord(event2_prepared,0));
    CHECK_CUSPARSE(hipsparseSetStream(this->cudaSpMatrix->cuSparseHandle,0));
    
    // execute SpMV
    //CHECK_CUSPARSE(hipsparseSpMV(handle, trans, &alpha, matA, vecX, &beta, vecY, HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, d_buffer));
    CHECK_CUSPARSE(hipsparseSpMV(this->cudaSpMatrix->cuSparseHandle, op_transpose, &alpha, this->cudaSpMatrix->cuSparseMatrix, vecX, &beta, vecY, cudaType<val_t>::kind, algorithm, d_buffer));

    CHECK_CUDA(hipEventRecord(event3_spmv,0));

    //Copy result
    CHECK_CUDA(hipMemcpy(result_data, result_d, m * sizeof(mxSingle), hipMemcpyDeviceToHost));

    CHECK_CUDA(hipEventRecord(event4_result,0));

    //Synchronize - should be done with memcpy anywas
    //CHECK_CUDA(hipDeviceSynchronize()); 
    
    //Free buffers etc.
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX));
    CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY));
    //CHECK_CUSPARSE(hipsparseDestroy(cuSparseHandle)); //NOw done in destructor
    if (d_buffer) CHECK_CUDA(hipFree(d_buffer));
    CHECK_CUDA(hipFree(result_d));
    CHECK_CUDA(hipFree(vals_d));

    CHECK_CUDA(hipEventRecord(event5_cleanup,0));
    CHECK_CUDA(hipEventSynchronize(event5_cleanup));

    float time2_prepared, time3_spmv, time4_result, time5_cleanup, timeAll;
    CHECK_CUDA(hipEventElapsedTime(&time2_prepared,event1_start,event2_prepared));
    CHECK_CUDA(hipEventElapsedTime(&time3_spmv,event2_prepared,event3_spmv));
    CHECK_CUDA(hipEventElapsedTime(&time4_result,event3_spmv,event4_result));
    CHECK_CUDA(hipEventElapsedTime(&time5_cleanup,event4_result,event5_cleanup));
    CHECK_CUDA(hipEventElapsedTime(&timeAll,event1_start,event5_cleanup));

    #ifdef CUDA_MEX_PERFANA
        mexPrintf("GPU Times (All=%3.1f ms):\n\tprepare: %3.1f ms\n\tSpMv: %3.1f ms\n\tcopy result: %3.1f ms\n\tcleanup: %3.1f ms\n",
            timeAll,time2_prepared, time3_spmv, time4_result, time5_cleanup);
    #endif

    //return the bare array
    return result;
}

sparseSingleGPU::sparseSingleGPU(std::shared_ptr<sparseSingleGPUdata> cudaSpMatrix_) {
    this->cudaSpMatrix = cudaSpMatrix_;
}

sparseSingleGPU* sparseSingleGPU::transpose() const {
    sparseSingleGPU* transposedCopy = new sparseSingleGPU(this->cudaSpMatrix);
    transposedCopy->transposed = !this->transposed;
    
    return transposedCopy;
}

mxArray* sparseSingleGPU::vecTimes(const mxSingle* vals,mwSize n, bool transposeInPlace) const 
{
    //Result Dimension: We expect a row vector from x*A (x is a row vector)
    mwSize resultDim[2];
    resultDim[0] = 1;
    resultDim[1] = this->getCols();
    
    //Call A*x with transposed matrix. We use try catch to be able to perform the transpose reverse even when the operation fails
    mxArray* result = this->timesVec(vals,n,true);
    
    //Transpose result efficiently
    int status = mxSetDimensions(result,resultDim,2);      

    return result;
}